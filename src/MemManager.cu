#include <cstdlib>
#include"../include/MemManager.cuh"
using namespace std;
namespace DNN_FrameWork
{
    MemManager::MemManager(size_t size):cpu_ptr(NULL), gpu_ptr(NULL) ,size(size) , state(UNINITIALIZED) {}
    MemManager::MemManager():cpu_ptr(NULL), gpu_ptr(NULL) ,size(0) , state(UNINITIALIZED) {}

    MemManager::~MemManager(){
        if(this->state == IN_CPU)
            free(this->cpu_ptr);
        else if (this->state == IN_GPU)
            hipFree(this->gpu_ptr);

        
    }
    const void * MemManager::gpu_data(){
        to_gpu();
        state = IN_GPU;
        return (const void*)this->gpu_ptr;
    }

    const void * MemManager::cpu_data(){
        to_cpu();
        state = IN_CPU;
        return (const void*)this->cpu_ptr;
    }
    void MemManager::freeGPU(){
        hipFree(this->gpu_ptr);
    }
    void MemManager::freeCPU(){
        free(this->cpu_ptr);
    }
    void MemManager::to_gpu(){
        switch (state)
        {
        case UNINITIALIZED:
            hipMalloc((void**)&this->gpu_ptr,size);
            break;
        case IN_CPU:
            if(gpu_ptr==NULL){
                hipMalloc((void**)&this->gpu_ptr,this->size);
            }
            hipMemcpy(gpu_ptr,cpu_ptr,size,hipMemcpyHostToDevice);
            break;
        case IN_GPU:
            break;
        default:
            break;
        }

    }
    void MemManager::to_cpu(){
        switch (state)
        {
        case UNINITIALIZED:
            this->cpu_ptr = malloc(this->size);
            break;
        case IN_CPU:
            break;
        case IN_GPU:
            if(cpu_ptr==NULL){
                this->cpu_ptr=malloc(size);
            }
            hipMemcpy(cpu_ptr,gpu_ptr,size,hipMemcpyDeviceToHost);
            break;
        default:
            break;
        }
        
    }
    void * MemManager::mutable_gpu_data(){
        to_gpu();
        state = IN_GPU;
        return (void*)this->gpu_ptr;
    }
    void * MemManager::mutable_cpu_data(){
        to_cpu();
        state = IN_CPU;
        return (void*)this->cpu_ptr;
    }
    void MemManager::set_cpu_data(void* data){
        if(cpu_ptr!=NULL){
            free(cpu_ptr);
        }
        cpu_ptr = data;
        state = IN_CPU;
    }
    void MemManager::set_gpu_data(void* data){
        if(gpu_ptr!=NULL){
            hipFree(gpu_ptr);
        }
        gpu_ptr = data;
        state = IN_GPU;
    }
    void MemManager::DNNcudaMemCpyToHost(void*HostPtr,size_t bytes,void * devPtr){
        hipMemcpy(HostPtr,devPtr,bytes,hipMemcpyDeviceToHost);
    }
    void MemManager::DNNcudaMemCpyToDevice(void * devPtr,size_t bytes,void*HostPtr){
        hipMemcpy(devPtr,HostPtr,bytes,hipMemcpyHostToDevice);
    }

    void MemManager::DNNcudaMemCpyDefault(void * devPtr,size_t bytes,void*HostPtr){
        hipMemcpy(devPtr,HostPtr,bytes,hipMemcpyDefault);
    }
    
} // namespace DNN_FrameWork

